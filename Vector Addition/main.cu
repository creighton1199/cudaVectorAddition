#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <time.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;
    time_t t;

    float* Ad, *Bd, *Cd; 


    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }
    
    /* Intializes random number generator */
    srand((unsigned) time(&t));    
    

    float* A_h = (float*) malloc(n*sizeof(float));
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc(n*sizeof(float));
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc(n*sizeof(float));

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    if (hipSuccess!=hipMalloc((void**)&Ad, n*sizeof(float)))
    {
        printf("Error in memory allocation/n");
        exit(-1);
    }
    if (hipSuccess!=hipMalloc((void**)&Bd, n*sizeof(float)))
    {
        printf("Error allocating memory\n");
        exit(-1);
    }
    if (hipSuccess!=hipMalloc((void**)&Cd, n*sizeof(float)))
    {
        printf("Error allocating memory\n");
        exit(-1);
    }










    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    int size=n*sizeof(float);

    if (hipSuccess !=  hipMemcpy(Ad,A_h, size, hipMemcpyHostToDevice))
    {
        printf("Error copying memory to data\n");
        exit(-1);
    }
    if (hipSuccess != hipMemcpy(Bd,B_h, size, hipMemcpyHostToDevice))
    {
        printf("Error copying memory to data\n");
        exit(-1);
    }




    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    dim3 dimBlock(16,16,1);
    dim3 dimGrid(ceil(n/16),1,1);

    vecAddKernel<<<dimGrid,dimBlock>>>(Ad,Bd, Cd, n);



    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    
    if (hipSuccess != hipMemcpy(C_h,Cd, size, hipMemcpyDeviceToHost))
    {
        printf("Error copying data to host");
        exit(-1);
    }
    


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE

    if (hipSuccess != hipFree(Ad))
    {
        printf("Error releasing cuda");
        exit(-1);
    }
    if (hipSuccess != hipFree(Bd))
    {
        printf("Error releasing cuda");
        exit(-1);
    }
    if (hipSuccess != hipFree(Cd))
    {
        printf("Error releasing cuda");
        exit(-1);
    }


    return 0;

}

